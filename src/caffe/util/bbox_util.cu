#include "hip/hip_runtime.h"
#include <algorithm>
#include <functional>
#include <map>
#include <vector>

#include "thrust/functional.h"
#include "thrust/sort.h"

#include "caffe/common.hpp"
#include "caffe/util/bbox_util.hpp"

namespace caffe {

template <typename Dtype>
__host__ __device__ Dtype BBoxSizeGPU(const Dtype* bbox,
    const bool normalized) {
  if (bbox[2] < bbox[0] || bbox[3] < bbox[1]) {
    // If bbox is invalid (e.g. xmax < xmin or ymax < ymin), return 0.
    return Dtype(0.);
  } else {
    const Dtype width = bbox[2] - bbox[0];
    const Dtype height = bbox[3] - bbox[1];
    if (normalized) {
      return width * height;
    } else {
      // If bbox is not within range [0, 1].
      return (width + 1) * (height + 1);
    }
  }
}

template __host__ __device__ float BBoxSizeGPU(const float* bbox,
    const bool normalized);
template __host__ __device__ double BBoxSizeGPU(const double* bbox,
    const bool normalized);
//**********************************************************************
template <typename Dtype>
__host__ __device__ Dtype PBoxSizeGPU(const Dtype* pbox,
    const bool normalized) {
	
  Dtype ltx, lty, rbx, rby, lbx, lby, rtx,rty;
    ltx = pbox[0]; lty = pbox[1];
    rbx = pbox[4]; rby = pbox[5];
    lbx = pbox[2]; lby = pbox[3];
    rtx = pbox[6];  rty = pbox[7];
    Dtype acad, bcbd, cacb, dadb;
    acad = ((rtx - ltx)*(lby-lty) - (lbx - ltx)*(rty - lty));
    bcbd = ((rtx - rbx)*(lby-rby)- (lbx-rbx)*(rty-rby));
    cacb = ((ltx - rtx)*(rby-rty)-(rbx-rtx)*(lty-rty));
    dadb = ((ltx-lbx)*(rby-lby)- (rbx-lbx)*(lty-lby));
	if (acad*bcbd>0 || cacb*dadb>0) {
	// If pbox is invalid (e.g. xmax < xmin or ymax < ymin), return 0.
		return 0;
	}
  else {
			const Dtype ltrbx = pbox[0] - pbox[4];
			const Dtype ltrby = pbox[1] - pbox[5];
			const Dtype lbrtx = pbox[6] - pbox[2];
			const Dtype lbrty = pbox[7] - pbox[3];
			//float width = bbox.xmax() - bbox.xmin();
			//float height = bbox.ymax() - bbox.ymin();
			if (normalized) {

				Dtype ltrb = sqrtf(powf(ltrbx, 2.f) + powf(ltrby, 2.f));
				Dtype lbrt = sqrtf(powf(lbrtx, 2.f) + powf(lbrty, 2.f));
				Dtype alpha = fabsf(sinf(atan2f(ltrbx, ltrby) - atan2f(lbrtx, lbrty)));
				return ltrb * lbrt*alpha / 2.f;
			}
			else {
				// If pbox is not within range [0, 1].
				
				Dtype ltrb = sqrtf(powf(ltrbx+1, 2.f) + powf(ltrby+1, 2.f));
				Dtype lbrt = sqrtf(powf(lbrtx+1, 2.f) + powf(lbrty+1, 2.f));
				Dtype alpha = fabsf(sinf(atan2f(ltrbx+1, ltrby+1) - atan2f(lbrtx+1, lbrty+1)));
				return ltrb * lbrt*alpha / 2.f;
    }
  }
} 	

template <typename Dtype>
__host__ __device__ Dtype JaccardOverlapGPU(const Dtype* bbox1,
    const Dtype* bbox2) {
  if (bbox2[0] > bbox1[2] || bbox2[2] < bbox1[0] ||
      bbox2[1] > bbox1[3] || bbox2[3] < bbox1[1]) {
    return Dtype(0.);
  } else {
    const Dtype inter_xmin = max(bbox1[0], bbox2[0]);
    const Dtype inter_ymin = max(bbox1[1], bbox2[1]);
    const Dtype inter_xmax = min(bbox1[2], bbox2[2]);
    const Dtype inter_ymax = min(bbox1[3], bbox2[3]);

    const Dtype inter_width = inter_xmax - inter_xmin;
    const Dtype inter_height = inter_ymax - inter_ymin;
    const Dtype inter_size = inter_width * inter_height;

    const Dtype bbox1_size = BBoxSizeGPU(bbox1);
    const Dtype bbox2_size = BBoxSizeGPU(bbox2);

    return inter_size / (bbox1_size + bbox2_size - inter_size);
  }
}

template __host__ __device__ float JaccardOverlapGPU(const float* bbox1,
    const float* bbox2);
template __host__ __device__ double JaccardOverlapGPU(const double* bbox1,
    const double* bbox2);

//****************************************************************	
template <typename Dtype>
__host__ __device__ Dtype JaccardOverlapPboxGPU(const Dtype* pbox1,
    const Dtype* pbox2) {
  /* if (bbox2[0] > bbox1[2] || bbox2[2] < bbox1[0] ||
      bbox2[1] > bbox1[3] || bbox2[3] < bbox1[1]) {
    return Dtype(0.);
  } else { */
	  
	 // float d1, d2, d3, d4;
	const Dtype d1 = sqrtf((powf((pbox1[0] - pbox2[0]), 2.f) + powf((pbox1[1] - pbox2[1]), 2.f)));
	const Dtype d2 = sqrtf((powf((pbox1[2] - pbox2[2]), 2.f) + powf((pbox1[3] - pbox2[3]), 2.f)));
	const Dtype d3 = sqrtf((powf((pbox1[4] - pbox2[4]), 2.f) + powf((pbox1[5] - pbox2[5]), 2.f)));
	const Dtype d4 = sqrtf((powf((pbox1[6] - pbox2[6]), 2.f) + powf((pbox1[7] - pbox2[7]), 2.f)));

	const Dtype d = max(max(max(d1, d2), d3), d4);

	const Dtype D1 = sqrtf((powf((pbox1[0] - pbox1[2]), 2.f) + powf((pbox1[1] - pbox1[3]), 2.f)));
	const Dtype D2 = sqrtf((powf((pbox1[2] - pbox1[4]), 2.f) + powf((pbox1[3] - pbox1[5]), 2.f)));
	const Dtype D3 = sqrtf((powf((pbox1[4] - pbox1[6]), 2.f) + powf((pbox1[5] - pbox1[7]), 2.f)));
	const Dtype D4 = sqrtf((powf((pbox1[6] - pbox1[0]), 2.f) + powf((pbox1[7] - pbox1[1]), 2.f)));

	const Dtype D = min(min(min(D1, D2), D3), D4);

	return 1.f - expf(-D / d);
  //}
}

template __host__ __device__ float JaccardOverlapPboxGPU(const float* pbox1,
    const float* pbox2);
template __host__ __device__ double JaccardOverlapPboxGPU(const double* pbox1,
    const double* pbox2);	

template <typename Dtype>
__device__ Dtype Min(const Dtype x, const Dtype y) {
  return x < y ? x : y;
}

template <typename Dtype>
__device__ Dtype Max(const Dtype x, const Dtype y) {
  return x > y ? x : y;
}

/*template <typename Dtype>*/
/*__device__ Dtype Pow(const Dtype x, const Dtype n) {*/
    /*Dtype result = Dtype(1.);*/
    /*for (int i = 0;i<n; i++) {*/
        /*result = x*result;*/
    /*}*/
    /*return result;*/
/*}*/

template <typename Dtype>
__device__ void ClipBBoxGPU(const Dtype* bbox, Dtype* clip_bbox) {
  for (int i = 0; i < 4; ++i) {
    clip_bbox[i] = Max(Min(bbox[i], Dtype(1.)), Dtype(0.));
  }
}

template __device__ void ClipBBoxGPU(const float* bbox, float* clip_bbox);
template __device__ void ClipBBoxGPU(const double* bbox, double* clip_bbox);

//***************************************************************************
template <typename Dtype>
__device__ void ClipPBoxGPU(const Dtype* pbox, Dtype* clip_pbox) {
  for (int i = 0; i < 8; ++i) {
    clip_pbox[i] = Max(Min(pbox[i], Dtype(1.)), Dtype(0.));
  }
}

template __device__ void ClipPBoxGPU(const float* pbox, float* clip_pbox);
template __device__ void ClipPBoxGPU(const double* pbox, double* clip_pbox);


template <typename Dtype>
__global__ void DecodeBBoxesKernel(const int nthreads,
          const Dtype* loc_data, const Dtype* prior_data,
          const CodeType code_type, const bool variance_encoded_in_target,
          const int num_priors, const bool share_location,
          const int num_loc_classes, const int background_label_id,
          const bool clip_bbox, Dtype* bbox_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int i = index % 4;
    const int c = (index / 4) % num_loc_classes;
    const int d = (index / 4 / num_loc_classes) % num_priors;
    if (!share_location && c == background_label_id) {
      // Ignore background class if not share_location.
      return;
    }
    const int pi = d * 4;
    const int vi = pi + num_priors * 4;
    if (code_type == PriorBoxParameter_CodeType_CORNER) {
      if (variance_encoded_in_target) {
        // variance is encoded in target, we simply need to add the offset
        // predictions.
        bbox_data[index] = prior_data[pi + i] + loc_data[index];
      } else {
        // variance is encoded in bbox, we need to scale the offset accordingly.
        bbox_data[index] =
          prior_data[pi + i] + loc_data[index] * prior_data[vi + i];
      }
    } else if (code_type == PriorBoxParameter_CodeType_CENTER_SIZE) {
      const Dtype p_xmin = prior_data[pi];
      const Dtype p_ymin = prior_data[pi + 1];
      const Dtype p_xmax = prior_data[pi + 2];
      const Dtype p_ymax = prior_data[pi + 3];
      const Dtype prior_width = p_xmax - p_xmin;
      const Dtype prior_height = p_ymax - p_ymin;
      const Dtype prior_center_x = (p_xmin + p_xmax) / 2.;
      const Dtype prior_center_y = (p_ymin + p_ymax) / 2.;

      const Dtype xmin = loc_data[index - i];
      const Dtype ymin = loc_data[index - i + 1];
      const Dtype xmax = loc_data[index - i + 2];
      const Dtype ymax = loc_data[index - i + 3];

      Dtype decode_bbox_center_x, decode_bbox_center_y;
      Dtype decode_bbox_width, decode_bbox_height;
      if (variance_encoded_in_target) {
        // variance is encoded in target, we simply need to retore the offset
        // predictions.
        decode_bbox_center_x = xmin * prior_width + prior_center_x;
        decode_bbox_center_y = ymin * prior_height + prior_center_y;
        decode_bbox_width = exp(xmax) * prior_width;
        decode_bbox_height = exp(ymax) * prior_height;
      } else {
        // variance is encoded in bbox, we need to scale the offset accordingly.
        decode_bbox_center_x =
          prior_data[vi] * xmin * prior_width + prior_center_x;
        decode_bbox_center_y =
          prior_data[vi + 1] * ymin * prior_height + prior_center_y;
        decode_bbox_width =
          exp(prior_data[vi + 2] * xmax) * prior_width;
        decode_bbox_height =
          exp(prior_data[vi + 3] * ymax) * prior_height;
      }

      switch (i) {
        case 0:
          bbox_data[index] = decode_bbox_center_x - decode_bbox_width / 2.;
          break;
        case 1:
          bbox_data[index] = decode_bbox_center_y - decode_bbox_height / 2.;
          break;
        case 2:
          bbox_data[index] = decode_bbox_center_x + decode_bbox_width / 2.;
          break;
        case 3:
          bbox_data[index] = decode_bbox_center_y + decode_bbox_height / 2.;
          break;
      }
    } else if (code_type == PriorBoxParameter_CodeType_CORNER_SIZE) {
      const Dtype p_xmin = prior_data[pi];
      const Dtype p_ymin = prior_data[pi + 1];
      const Dtype p_xmax = prior_data[pi + 2];
      const Dtype p_ymax = prior_data[pi + 3];
      const Dtype prior_width = p_xmax - p_xmin;
      const Dtype prior_height = p_ymax - p_ymin;
      Dtype p_size;
      if (i == 0 || i == 2) {
        p_size = prior_width;
      } else {
        p_size = prior_height;
      }
      if (variance_encoded_in_target) {
        // variance is encoded in target, we simply need to add the offset
        // predictions.
        bbox_data[index] = prior_data[pi + i] + loc_data[index] * p_size;
      } else {
        // variance is encoded in bbox, we need to scale the offset accordingly.
        bbox_data[index] =
          prior_data[pi + i] + loc_data[index] * prior_data[vi + i] * p_size;
      }
    } else {
      // Unknown code type.
    }
    if (clip_bbox) {
      bbox_data[index] = max(min(bbox_data[index], Dtype(1.)), Dtype(0.));
    }
  }
}

template <typename Dtype>
void DecodeBBoxesGPU(const int nthreads,
          const Dtype* loc_data, const Dtype* prior_data,
          const CodeType code_type, const bool variance_encoded_in_target,
          const int num_priors, const bool share_location,
          const int num_loc_classes, const int background_label_id,
          const bool clip_bbox, Dtype* bbox_data) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  DecodeBBoxesKernel<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, loc_data, prior_data, code_type,
      variance_encoded_in_target, num_priors, share_location, num_loc_classes,
      background_label_id, clip_bbox, bbox_data);
  CUDA_POST_KERNEL_CHECK;
}

template void DecodeBBoxesGPU(const int nthreads,
          const float* loc_data, const float* prior_data,
          const CodeType code_type, const bool variance_encoded_in_target,
          const int num_priors, const bool share_location,
          const int num_loc_classes, const int background_label_id,
          const bool clip_bbox, float* bbox_data);
template void DecodeBBoxesGPU(const int nthreads,
          const double* loc_data, const double* prior_data,
          const CodeType code_type, const bool variance_encoded_in_target,
          const int num_priors, const bool share_location,
          const int num_loc_classes, const int background_label_id,
          const bool clip_bbox, double* bbox_data);
		  
//*********************************************************************************
template <typename Dtype>
__global__ void DecodePBoxesKernel(const int nthreads,
          const Dtype* loc_data, const Dtype* prior_data,
          const PCodeType code_type, const bool variance_encoded_in_target,
          const int num_priors, const bool share_location,
          const int num_loc_classes, const int background_label_id,
          const bool clip_pbox, Dtype* pbox_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int i = index % 4;
    const int c = (index / 8) % num_loc_classes;
    const int d = (index / 8 / num_loc_classes) % num_priors;
    if (!share_location && c == background_label_id) {
      // Ignore background class if not share_location.
      return;
    }
    const int pi = d * 8;
    const int vi = pi + num_priors * 8;
    if (code_type == PriorPBoxParameter_CodeType_CORNER) {
      if (variance_encoded_in_target) {
        // variance is encoded in target, we simply need to add the offset
        // predictions.
        pbox_data[index] = prior_data[pi + i] + loc_data[index];
      } else {
        // variance is encoded in pbox, we need to scale the offset accordingly.
        pbox_data[index] =
          prior_data[pi + i] + loc_data[index] * prior_data[vi + i];
      }
    } /* else if (code_type == PriorPBoxParameter_CodeType_CENTER_SIZE) {
      const Dtype p_xmin = prior_data[pi];
      const Dtype p_ymin = prior_data[pi + 1];
      const Dtype p_xmax = prior_data[pi + 2];
      const Dtype p_ymax = prior_data[pi + 3];
      const Dtype prior_width = p_xmax - p_xmin;
      const Dtype prior_height = p_ymax - p_ymin;
      const Dtype prior_center_x = (p_xmin + p_xmax) / 2.;
      const Dtype prior_center_y = (p_ymin + p_ymax) / 2.;

      const Dtype xmin = loc_data[index - i];
      const Dtype ymin = loc_data[index - i + 1];
      const Dtype xmax = loc_data[index - i + 2];
      const Dtype ymax = loc_data[index - i + 3];

      Dtype decode_bbox_center_x, decode_bbox_center_y;
      Dtype decode_bbox_width, decode_bbox_height;
      if (variance_encoded_in_target) {
        // variance is encoded in target, we simply need to retore the offset
        // predictions.
        decode_bbox_center_x = xmin * prior_width + prior_center_x;
        decode_bbox_center_y = ymin * prior_height + prior_center_y;
        decode_bbox_width = exp(xmax) * prior_width;
        decode_bbox_height = exp(ymax) * prior_height;
      } else {
        // variance is encoded in bbox, we need to scale the offset accordingly.
        decode_bbox_center_x =
          prior_data[vi] * xmin * prior_width + prior_center_x;
        decode_bbox_center_y =
          prior_data[vi + 1] * ymin * prior_height + prior_center_y;
        decode_bbox_width =
          exp(prior_data[vi + 2] * xmax) * prior_width;
        decode_bbox_height =
          exp(prior_data[vi + 3] * ymax) * prior_height;
      }

      switch (i) {
        case 0:
          bbox_data[index] = decode_bbox_center_x - decode_bbox_width / 2.;
          break;
        case 1:
          bbox_data[index] = decode_bbox_center_y - decode_bbox_height / 2.;
          break;
        case 2:
          bbox_data[index] = decode_bbox_center_x + decode_bbox_width / 2.;
          break;
        case 3:
          bbox_data[index] = decode_bbox_center_y + decode_bbox_height / 2.;
          break;
      }
    } else if (code_type == PriorBoxParameter_CodeType_CORNER_SIZE) {
      const Dtype p_xmin = prior_data[pi];
      const Dtype p_ymin = prior_data[pi + 1];
      const Dtype p_xmax = prior_data[pi + 2];
      const Dtype p_ymax = prior_data[pi + 3];
      const Dtype prior_width = p_xmax - p_xmin;
      const Dtype prior_height = p_ymax - p_ymin;
      Dtype p_size;
      if (i == 0 || i == 2) {
        p_size = prior_width;
      } else {
        p_size = prior_height;
      }
      if (variance_encoded_in_target) {
        // variance is encoded in target, we simply need to add the offset
        // predictions.
        bbox_data[index] = prior_data[pi + i] + loc_data[index] * p_size;
      } else {
        // variance is encoded in bbox, we need to scale the offset accordingly.
        bbox_data[index] =
          prior_data[pi + i] + loc_data[index] * prior_data[vi + i] * p_size;
      }
    }  */
	else {
      // Unknown code type.
    }
    if (clip_pbox) {
      pbox_data[index] = max(min(pbox_data[index], Dtype(1.)), Dtype(0.));
    }
  }
}

template <typename Dtype>
void DecodePBoxesGPU(const int nthreads,
          const Dtype* loc_data, const Dtype* prior_data,
          const PCodeType code_type, const bool variance_encoded_in_target,
          const int num_priors, const bool share_location,
          const int num_loc_classes, const int background_label_id,
          const bool clip_pbox, Dtype* pbox_data) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  DecodePBoxesKernel<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, loc_data, prior_data, code_type,
      variance_encoded_in_target, num_priors, share_location, num_loc_classes,
      background_label_id, clip_pbox, pbox_data);
  CUDA_POST_KERNEL_CHECK;
}

template void DecodePBoxesGPU(const int nthreads,
          const float* loc_data, const float* prior_data,
          const PCodeType code_type, const bool variance_encoded_in_target,
          const int num_priors, const bool share_location,
          const int num_loc_classes, const int background_label_id,
          const bool clip_pbox, float* pbox_data);
template void DecodePBoxesGPU(const int nthreads,
          const double* loc_data, const double* prior_data,
          const PCodeType code_type, const bool variance_encoded_in_target,
          const int num_priors, const bool share_location,
          const int num_loc_classes, const int background_label_id,
          const bool clip_pbox, double* pbox_data);
		  

template <typename Dtype>
__global__ void PermuteDataKernel(const int nthreads,
          const Dtype* data, const int num_classes, const int num_data,
          const int num_dim, Dtype* new_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int i = index % num_dim;
    const int c = (index / num_dim) % num_classes;
    const int d = (index / num_dim / num_classes) % num_data;
    const int n = index / num_dim / num_classes / num_data;
    const int new_index = ((n * num_classes + c) * num_data + d) * num_dim + i;
    new_data[new_index] = data[index];
  }
}

template <typename Dtype>
void PermuteDataGPU(const int nthreads,
          const Dtype* data, const int num_classes, const int num_data,
          const int num_dim, Dtype* new_data) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  PermuteDataKernel<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, data, num_classes, num_data,
      num_dim, new_data);
  CUDA_POST_KERNEL_CHECK;
}

template void PermuteDataGPU(const int nthreads,
          const float* data, const int num_classes, const int num_data,
          const int num_dim, float* new_data);
template void PermuteDataGPU(const int nthreads,
          const double* data, const int num_classes, const int num_data,
          const int num_dim, double* new_data);

template <typename Dtype>
__global__ void kernel_channel_max(const int num, const int channels,
    const int spatial_dim, const Dtype* data, Dtype* out) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    Dtype maxval = -FLT_MAX;
    for (int c = 0; c < channels; ++c) {
      maxval = max(data[(n * channels + c) * spatial_dim + s], maxval);
    }
    out[index] = maxval;
  }
}

template <typename Dtype>
__global__ void kernel_channel_subtract(const int count,
    const int num, const int channels,
    const int spatial_dim, const Dtype* channel_data, const Dtype* channel_max,
    Dtype* data) {
  CUDA_KERNEL_LOOP(index, count) {
    int n = index / channels / spatial_dim;
    int s = index % spatial_dim;
    data[index] = channel_data[index] - channel_max[n * spatial_dim + s];
  }
}

template <typename Dtype>
__global__ void kernel_exp(const int count, const Dtype* data, Dtype* out) {
  CUDA_KERNEL_LOOP(index, count) {
    out[index] = exp(data[index]);
  }
}

template <typename Dtype>
__global__ void kernel_channel_sum(const int num, const int channels,
    const int spatial_dim, const Dtype* data, Dtype* channel_sum) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    Dtype sum = 0;
    for (int c = 0; c < channels; ++c) {
      sum += data[(n * channels + c) * spatial_dim + s];
    }
    channel_sum[index] = sum;
  }
}

template <typename Dtype>
__global__ void kernel_channel_div(const int count,
    const int num, const int channels,
    const int spatial_dim, const Dtype* channel_sum, Dtype* data) {
  CUDA_KERNEL_LOOP(index, count) {
    int n = index / channels / spatial_dim;
    int s = index % spatial_dim;
    data[index] /= channel_sum[n * spatial_dim + s];
  }
}

template <typename Dtype>
void SoftMaxGPU(const Dtype* data, const int outer_num,
    const int channels, const int inner_num, Dtype* prob) {
  vector<int> shape(4, 1);
  shape[0] = outer_num;
  shape[1] = channels;
  shape[2] = inner_num;
  Blob<Dtype> scale(shape);
  Dtype* scale_data = scale.mutable_gpu_data();
  int count = outer_num * channels * inner_num;
  // We need to subtract the max to avoid numerical issues, compute the exp,
  // and then normalize.
  // compute max
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_max<Dtype><<<CAFFE_GET_BLOCKS(outer_num * inner_num),
      CAFFE_CUDA_NUM_THREADS>>>(outer_num, channels, inner_num, data,
      scale_data);
  // subtract
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_subtract<Dtype><<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(count, outer_num, channels, inner_num,
      data, scale_data, prob);
  // exponentiate
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_exp<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, prob, prob);
  // sum after exp
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_sum<Dtype><<<CAFFE_GET_BLOCKS(outer_num * inner_num),
      CAFFE_CUDA_NUM_THREADS>>>(outer_num, channels, inner_num, prob,
      scale_data);
  // divide
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_div<Dtype><<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(count, outer_num, channels, inner_num,
      scale_data, prob);
}

template void SoftMaxGPU(const float* data, const int outer_num,
    const int channels, const int inner_num, float* prob);
template void SoftMaxGPU(const double* data, const int outer_num,
    const int channels, const int inner_num, double* prob);

	
template <typename Dtype>
__global__ void ComputeOverlappedKernel(const int nthreads,
          const Dtype* bbox_data, const int num_bboxes, const int num_classes,
          const Dtype overlap_threshold, bool* overlapped_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int j = index % num_bboxes;
    const int i = (index / num_bboxes) % num_bboxes;
    if (i == j) {
      // Ignore same bbox.
      return;
    }
    const int c = (index / num_bboxes / num_bboxes) % num_classes;
    const int n = index / num_bboxes / num_bboxes / num_classes;
    // Compute overlap between i-th bbox and j-th bbox.
    const int start_loc_i = ((n * num_bboxes + i) * num_classes + c) * 4;
    const int start_loc_j = ((n * num_bboxes + j) * num_classes + c) * 4;
    const Dtype overlap = JaccardOverlapGPU<Dtype>(bbox_data + start_loc_i,
        bbox_data + start_loc_j);
    if (overlap > overlap_threshold) {
      overlapped_data[index] = true;
    }
  }
}

template <typename Dtype>
void ComputeOverlappedGPU(const int nthreads,
          const Dtype* bbox_data, const int num_bboxes, const int num_classes,
          const Dtype overlap_threshold, bool* overlapped_data) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  ComputeOverlappedKernel<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, bbox_data, num_bboxes, num_classes,
      overlap_threshold, overlapped_data);
  CUDA_POST_KERNEL_CHECK;
}

template void ComputeOverlappedGPU(const int nthreads,
          const float* bbox_data, const int num_bboxes, const int num_classes,
          const float overlap_threshold, bool* overlapped_data);
template void ComputeOverlappedGPU(const int nthreads,
          const double* bbox_data, const int num_bboxes, const int num_classes,
          const double overlap_threshold, bool* overlapped_data);

//******************************************************************************
template <typename Dtype>
__global__ void ComputeOverlappedPboxKernel(const int nthreads,
          const Dtype* pbox_data, const int num_pboxes, const int num_classes,
          const Dtype overlap_threshold, bool* overlapped_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int j = index % num_pboxes;
    const int i = (index / num_pboxes) % num_pboxes;
    if (i == j) {
      // Ignore same pbox.
      return;
    }
    const int c = (index / num_pboxes / num_pboxes) % num_classes;
    const int n = index / num_pboxes / num_pboxes / num_classes;
    // Compute overlap between i-th pbox and j-th pbox.
    const int start_loc_i = ((n * num_pboxes + i) * num_classes + c) * 8;
    const int start_loc_j = ((n * num_pboxes + j) * num_classes + c) * 8;
    const Dtype overlap = JaccardOverlapPboxGPU<Dtype>(pbox_data + start_loc_i,
        pbox_data + start_loc_j);
    if (overlap > overlap_threshold) {
      overlapped_data[index] = true;
    }
  }
}

template <typename Dtype>
void ComputeOverlappedPboxGPU(const int nthreads,
          const Dtype* pbox_data, const int num_pboxes, const int num_classes,
          const Dtype overlap_threshold, bool* overlapped_data) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  ComputeOverlappedPboxKernel<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, pbox_data, num_pboxes, num_classes,
      overlap_threshold, overlapped_data);
  CUDA_POST_KERNEL_CHECK;
}

template void ComputeOverlappedPboxGPU(const int nthreads,
          const float* pbox_data, const int num_pboxes, const int num_classes,
          const float overlap_threshold, bool* overlapped_data);
template void ComputeOverlappedPboxGPU(const int nthreads,
          const double* pbox_data, const int num_pboxes, const int num_classes,
          const double overlap_threshold, bool* overlapped_data);		  

template <typename Dtype>
__global__ void ComputeOverlappedByIdxKernel(const int nthreads,
          const Dtype* bbox_data, const Dtype overlap_threshold,
          const int* idx, const int num_idx, bool* overlapped_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int j = index % num_idx;
    const int i = (index / num_idx);
    if (i == j) {
      // Ignore same bbox.
      return;
    }
    // Compute overlap between i-th bbox and j-th bbox.
    const int start_loc_i = idx[i] * 4;
    const int start_loc_j = idx[j] * 4;
    const Dtype overlap = JaccardOverlapGPU<Dtype>(bbox_data + start_loc_i,
        bbox_data + start_loc_j);
    if (overlap > overlap_threshold) {
      overlapped_data[index] = true;
    }
  }
}

template <typename Dtype>
void ComputeOverlappedByIdxGPU(const int nthreads,
          const Dtype* bbox_data, const Dtype overlap_threshold,
          const int* idx, const int num_idx, bool* overlapped_data) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  ComputeOverlappedByIdxKernel<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, bbox_data, overlap_threshold,
      idx, num_idx, overlapped_data);
  CUDA_POST_KERNEL_CHECK;
}

template void ComputeOverlappedByIdxGPU(const int nthreads,
          const float* bbox_data, const float overlap_threshold,
          const int* idx, const int num_idx, bool* overlapped_data);
template void ComputeOverlappedByIdxGPU(const int nthreads,
          const double* bbox_data, const double overlap_threshold,
          const int* idx, const int num_idx, bool* overlapped_data);
		  
template <typename Dtype>
__global__ void ComputeOverlappedPboxByIdxKernel(const int nthreads,
          const Dtype* pbox_data, const Dtype overlap_threshold,
          const int* idx, const int num_idx, bool* overlapped_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int j = index % num_idx;
    const int i = (index / num_idx);
    if (i == j) {
      // Ignore same pbox.
      return;
    }
    // Compute overlap between i-th pbox and j-th pbox.
    const int start_loc_i = idx[i] * 8;
    const int start_loc_j = idx[j] * 8;
    const Dtype overlap = JaccardOverlapPboxGPU<Dtype>(pbox_data + start_loc_i,
        pbox_data + start_loc_j);
    if (overlap > overlap_threshold) {
      overlapped_data[index] = true;
    }
  }
}

template <typename Dtype>
void ComputeOverlappedPboxByIdxGPU(const int nthreads,
          const Dtype* pbox_data, const Dtype overlap_threshold,
          const int* idx, const int num_idx, bool* overlapped_data) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  ComputeOverlappedPboxByIdxKernel<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, pbox_data, overlap_threshold,
      idx, num_idx, overlapped_data);
  CUDA_POST_KERNEL_CHECK;
}

template void ComputeOverlappedPboxByIdxGPU(const int nthreads,
          const float* pbox_data, const float overlap_threshold,
          const int* idx, const int num_idx, bool* overlapped_data);
template void ComputeOverlappedPboxByIdxGPU(const int nthreads,
          const double* pbox_data, const double overlap_threshold,
          const int* idx, const int num_idx, bool* overlapped_data);

/* *****************************************************************************
template <typename Dtype>
__global__ void ComputeOverlappedPboxKernel(const int nthreads,
          const Dtype* pbox_data, const int num_pboxes, const int num_classes,
          const Dtype overlap_threshold, bool* overlapped_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int j = index % num_pboxes;
    const int i = (index / num_pboxes) % num_pboxes;
    if (i == j) {
      // Ignore same bbox.
      return;
    }
    const int c = (index / num_pboxes / num_pboxes) % num_classes;
    const int n = index / num_pboxes / num_pboxes / num_classes;
    // Compute overlap between i-th bbox and j-th bbox.
    const int start_loc_i = ((n * num_bboxes + i) * num_classes + c) * 8;
    const int start_loc_j = ((n * num_bboxes + j) * num_classes + c) * 8;
    const Dtype overlap = JaccardOverlapPboxGPU<Dtype>(pbox_data + start_loc_i,
        pbox_data + start_loc_j);
    if (overlap > overlap_threshold) {
      overlapped_data[index] = true;
    }
  }
}

template <typename Dtype>
void ComputeOverlappedPboxGPU(const int nthreads,
          const Dtype* pbox_data, const int num_pboxes, const int num_classes,
          const Dtype overlap_threshold, bool* overlapped_data) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  ComputeOverlappedPboxKernel<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, pbox_data, num_pboxes, num_classes,
      overlap_threshold, overlapped_data);
  CUDA_POST_KERNEL_CHECK;
}

template void ComputeOverlappedPboxGPU(const int nthreads,
          const float* bbox_data, const int num_bboxes, const int num_classes,
          const float overlap_threshold, bool* overlapped_data);
template void ComputeOverlappedPboxGPU(const int nthreads,
          const double* bbox_data, const int num_bboxes, const int num_classes,
          const double overlap_threshold, bool* overlapped_data);

template <typename Dtype>
__global__ void ComputeOverlappedPboxByIdxKernel(const int nthreads,
          const Dtype* pbox_data, const Dtype overlap_threshold,
          const int* idx, const int num_idx, bool* overlapped_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int j = index % num_idx;
    const int i = (index / num_idx);
    if (i == j) {
      // Ignore same bbox.
      return;
    }
    // Compute overlap between i-th bbox and j-th bbox.
    const int start_loc_i = idx[i] * 8;
    const int start_loc_j = idx[j] * 8;
    const Dtype overlap = JaccardOverlapPboxGPU<Dtype>(pbox_data + start_loc_i,
        pbox_data + start_loc_j);
    if (overlap > overlap_threshold) {
      overlapped_data[index] = true;
    }
  }
}

template <typename Dtype>
void ComputeOverlappedPboxByIdxGPU(const int nthreads,
          const Dtype* pbox_data, const Dtype overlap_threshold,
          const int* idx, const int num_idx, bool* overlapped_data) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  ComputeOverlappedPboxByIdxKernel<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, pbox_data, overlap_threshold,
      idx, num_idx, overlapped_data);
  CUDA_POST_KERNEL_CHECK;
}

template void ComputeOverlappedPboxByIdxGPU(const int nthreads,
          const float* pbox_data, const float overlap_threshold,
          const int* idx, const int num_idx, bool* overlapped_data);
template void ComputeOverlappedPboxByIdxGPU(const int nthreads,
          const double* pbox_data, const double overlap_threshold,
          const int* idx, const int num_idx, bool* overlapped_data); */

template <typename Dtype>
void ApplyNMSGPU(const Dtype* bbox_data, const Dtype* conf_data,
          const int num_bboxes, const float confidence_threshold,
          const int top_k, const float nms_threshold, vector<int>* indices) {
  // Keep part of detections whose scores are higher than confidence threshold.
  vector<int> idx;
  vector<Dtype> confidences;
  for (int i = 0; i < num_bboxes; ++i) {
    if (conf_data[i] > confidence_threshold) {
      idx.push_back(i);
      confidences.push_back(conf_data[i]);
    }
  }
  int num_remain = confidences.size();
  if (num_remain == 0) {
    return;
  }
  // Sort detections based on score.
  thrust::sort_by_key(&confidences[0], &confidences[0] + num_remain, &idx[0],
      thrust::greater<Dtype>());
  if (top_k > -1 && top_k < num_remain) {
    num_remain = top_k;
  }

  // Compute overlap between remaining detections.
  Blob<int> idx_blob(1, 1, 1, num_remain);
  int* idx_data = idx_blob.mutable_cpu_data();
  std::copy(idx.begin(), idx.begin() + num_remain, idx_data);

  Blob<bool> overlapped(1, 1, num_remain, num_remain);
  const int total_bboxes = overlapped.count();
  bool* overlapped_data = overlapped.mutable_gpu_data();
  ComputeOverlappedByIdxGPU<Dtype>(total_bboxes, bbox_data, nms_threshold,
      idx_blob.gpu_data(), num_remain, overlapped_data);

  // Do non-maximum suppression based on overlapped results.
  const bool* overlapped_results = overlapped.cpu_data();
  vector<int> selected_indices;
  ApplyNMS(overlapped_results, num_remain, &selected_indices);

  // Put back the selected information.
  for (int i = 0; i < selected_indices.size(); ++i) {
    indices->push_back(idx[selected_indices[i]]);
  }
}

template
void ApplyNMSGPU(const float* bbox_data, const float* conf_data,
          const int num_bboxes, const float confidence_threshold,
          const int top_k, const float nms_threshold, vector<int>* indices);
template
void ApplyNMSGPU(const double* bbox_data, const double* conf_data,
          const int num_bboxes, const float confidence_threshold,
          const int top_k, const float nms_threshold, vector<int>* indices);
		  
//******************************************************************************
template <typename Dtype>
void ApplyPboxNMSGPU(const Dtype* pbox_data, const Dtype* conf_data,
          const int num_pboxes, const float confidence_threshold,
          const int top_k, const float nms_threshold, vector<int>* indices) {
  // Keep part of detections whose scores are higher than confidence threshold.
  vector<int> idx;
  vector<Dtype> confidences;
  for (int i = 0; i < num_pboxes; ++i) {
    if (conf_data[i] > confidence_threshold) {
      idx.push_back(i);
      confidences.push_back(conf_data[i]);
    }
  }
  int num_remain = confidences.size();
  if (num_remain == 0) {
    return;
  }
  // Sort detections based on score.
  thrust::sort_by_key(&confidences[0], &confidences[0] + num_remain, &idx[0],
      thrust::greater<Dtype>());
  if (top_k > -1 && top_k < num_remain) {
    num_remain = top_k;
  }

  // Compute overlap between remaining detections.
  Blob<int> idx_blob(1, 1, 1, num_remain);
  int* idx_data = idx_blob.mutable_cpu_data();
  std::copy(idx.begin(), idx.begin() + num_remain, idx_data);

  Blob<bool> overlapped(1, 1, num_remain, num_remain);
  const int total_pboxes = overlapped.count();
  bool* overlapped_data = overlapped.mutable_gpu_data();
  ComputeOverlappedPboxByIdxGPU<Dtype>(total_pboxes, pbox_data, nms_threshold,
      idx_blob.gpu_data(), num_remain, overlapped_data);

  // Do non-maximum suppression based on overlapped results.
  const bool* overlapped_results = overlapped.cpu_data();
  vector<int> selected_indices;
  ApplyPboxNMS(overlapped_results, num_remain, &selected_indices);

  // Put back the selected information.
  for (int i = 0; i < selected_indices.size(); ++i) {
    indices->push_back(idx[selected_indices[i]]);
  }
}

template
void ApplyPboxNMSGPU(const float* pbox_data, const float* conf_data,
          const int num_pboxes, const float confidence_threshold,
          const int top_k, const float nms_threshold, vector<int>* indices);
template
void ApplyPboxNMSGPU(const double* pbox_data, const double* conf_data,
          const int num_pboxes, const float confidence_threshold,
          const int top_k, const float nms_threshold, vector<int>* indices);

		
		
template <typename Dtype>
__global__ void GetDetectionsKernel(const int nthreads,
          const Dtype* bbox_data, const Dtype* conf_data, const int image_id,
          const int label, const int* indices, const bool clip_bbox,
          Dtype* detection_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int det_idx = indices[index];
    detection_data[index * 7] = image_id;
    detection_data[index * 7 + 1] = label;
    detection_data[index * 7 + 2] = conf_data[det_idx];
    if (clip_bbox) {
      ClipBBoxGPU(&(bbox_data[det_idx * 4]), &(detection_data[index * 7 + 3]));
    } else {
      for (int i = 0; i < 4; ++i) {
        detection_data[index * 7 + 3 + i] = bbox_data[det_idx * 4 + i];
      }
    }
  }
}

template <typename Dtype>
void GetDetectionsGPU(const Dtype* bbox_data, const Dtype* conf_data,
          const int image_id, const int label, const vector<int>& indices,
          const bool clip_bbox, Blob<Dtype>* detection_blob) {
  // Store selected indices in array.
  int num_det = indices.size();
  if (num_det == 0) {
    return;
  }
  Blob<int> idx_blob(1, 1, 1, num_det);
  int* idx_data = idx_blob.mutable_cpu_data();
  std::copy(indices.begin(), indices.end(), idx_data);
  // Prepare detection_blob.
  detection_blob->Reshape(1, 1, num_det, 7);
  Dtype* detection_data = detection_blob->mutable_gpu_data();
  // NOLINT_NEXT_LINE(whitespace/operators)
  GetDetectionsKernel<Dtype><<<CAFFE_GET_BLOCKS(num_det),
      CAFFE_CUDA_NUM_THREADS>>>(num_det, bbox_data, conf_data, image_id, label,
      idx_blob.gpu_data(), clip_bbox, detection_data);
  CUDA_POST_KERNEL_CHECK;
}

template void GetDetectionsGPU(const float* bbox_data, const float* conf_data,
          const int image_id, const int label, const vector<int>& indices,
          const bool clip_bbox, Blob<float>* detection_blob);
template void GetDetectionsGPU(const double* bbox_data, const double* conf_data,
          const int image_id, const int label, const vector<int>& indices,
          const bool clip_bbox, Blob<double>* detection_blob);
		  
//******************************************************************************
template <typename Dtype>
__global__ void GetPboxDetectionsKernel(const int nthreads,
          const Dtype* pbox_data, const Dtype* conf_data, const int image_id,
          const int label, const int* indices, const bool clip_pbox,
          Dtype* detection_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int det_idx = indices[index];
    detection_data[index * 11] = image_id;
    detection_data[index * 11 + 1] = label;
    detection_data[index * 11 + 2] = conf_data[det_idx];
    if (clip_pbox) {
      ClipBBoxGPU(&(pbox_data[det_idx * 8]), &(detection_data[index * 11 + 3]));
    } else {
      for (int i = 0; i < 8; ++i) {
        detection_data[index * 11 + 3 + i] = pbox_data[det_idx * 8 + i];
      }
    }
  }
}

template <typename Dtype>
void GetPboxDetectionsGPU(const Dtype* pbox_data, const Dtype* conf_data,
          const int image_id, const int label, const vector<int>& indices,
          const bool clip_pbox, Blob<Dtype>* detection_blob) {
  // Store selected indices in array.
  int num_det = indices.size();
  if (num_det == 0) {
    return;
  }
  Blob<int> idx_blob(1, 1, 1, num_det);
  int* idx_data = idx_blob.mutable_cpu_data();
  std::copy(indices.begin(), indices.end(), idx_data);
  // Prepare detection_blob.
  detection_blob->Reshape(1, 1, num_det, 11);
  Dtype* detection_data = detection_blob->mutable_gpu_data();
  // NOLINT_NEXT_LINE(whitespace/operators)
  GetPboxDetectionsKernel<Dtype><<<CAFFE_GET_BLOCKS(num_det),
      CAFFE_CUDA_NUM_THREADS>>>(num_det, pbox_data, conf_data, image_id, label,
      idx_blob.gpu_data(), clip_pbox, detection_data);
  CUDA_POST_KERNEL_CHECK;
}

template void GetPboxDetectionsGPU(const float* pbox_data, const float* conf_data,
          const int image_id, const int label, const vector<int>& indices,
          const bool clip_pbox, Blob<float>* detection_blob);
template void GetPboxDetectionsGPU(const double* pbox_data, const double* conf_data,
          const int image_id, const int label, const vector<int>& indices,
          const bool clip_pbox, Blob<double>* detection_blob);		  
		  
template <typename Dtype>
__global__ void ComputeConfLossKernel(const int nthreads,
    const Dtype* conf_data, const int num_preds_per_class,
    const int num_classes, const ConfLossType loss_type,
    const Dtype* match_data, Dtype* conf_loss_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int label = match_data[index];
    int num = index / num_preds_per_class;
    int p = index % num_preds_per_class;
    int start_idx = (num * num_preds_per_class + p) * num_classes;
    Dtype loss = 0;
    if (loss_type == MultiBoxLossParameter_ConfLossType_SOFTMAX) {
      // Compute softmax probability.
      Dtype prob = conf_data[start_idx + label];
      loss = -log(Max(prob, Dtype(FLT_MIN)));
    } else if (loss_type == MultiBoxLossParameter_ConfLossType_LOGISTIC) {
      int target = 0;
      for (int c = 0; c < num_classes; ++c) {
        if (c == label) {
          target = 1;
        } else {
          target = 0;
        }
        Dtype input = conf_data[start_idx + c];
        loss -= input * (target - (input >= 0)) -
          log(1 + exp(input - 2 * input * (input >= 0)));
      }
    }
    conf_loss_data[index] = loss;
  }
}

template <typename Dtype>
void ComputeConfLossGPU(const Blob<Dtype>& conf_blob, const int num,
      const int num_preds_per_class, const int num_classes,
      const int background_label_id, const ConfLossType loss_type,
      const vector<map<int, vector<int> > >& all_match_indices,
      const map<int, vector<NormalizedBBox> >& all_gt_bboxes,
      vector<vector<float> >* all_conf_loss) {
  CHECK_LT(background_label_id, num_classes);
  Blob<Dtype> match_blob(num, num_preds_per_class, 1, 1);
  Dtype* match_data = match_blob.mutable_cpu_data();
  for (int i = 0; i < num; ++i) {
    const map<int, vector<int> >& match_indices = all_match_indices[i];
    for (int p = 0; p < num_preds_per_class; ++p) {
      // Get the label index.
      int label = background_label_id;
      for (map<int, vector<int> >::const_iterator it =
           match_indices.begin(); it != match_indices.end(); ++it) {
        const vector<int>& match_index = it->second;
        CHECK_EQ(match_index.size(), num_preds_per_class);
        if (match_index[p] > -1) {
          CHECK(all_gt_bboxes.find(i) != all_gt_bboxes.end());
          const vector<NormalizedBBox>& gt_bboxes =
              all_gt_bboxes.find(i)->second;
          CHECK_LT(match_index[p], gt_bboxes.size());
          label = gt_bboxes[match_index[p]].label();
          CHECK_GE(label, 0);
          CHECK_NE(label, background_label_id);
          CHECK_LT(label, num_classes);
          // A prior can only be matched to one gt bbox.
          break;
        }
      }
      match_data[i * num_preds_per_class + p] = label;
    }
  }
  // Get probability data.
  const Dtype* conf_gpu_data = conf_blob.gpu_data();
  Blob<Dtype> prob_blob;
  prob_blob.ReshapeLike(conf_blob);
  if (loss_type == MultiBoxLossParameter_ConfLossType_SOFTMAX) {
    Dtype* prob_gpu_data = prob_blob.mutable_gpu_data();
    SoftMaxGPU(conf_blob.gpu_data(), num * num_preds_per_class, num_classes, 1,
        prob_gpu_data);
    conf_gpu_data = prob_blob.gpu_data();
  }
  // Compute the loss.
  Blob<Dtype> conf_loss_blob(num, num_preds_per_class, 1, 1);
  Dtype* conf_loss_gpu_data = conf_loss_blob.mutable_gpu_data();
  const int num_threads = num * num_preds_per_class;
  // NOLINT_NEXT_LINE(whitespace/operators)
  ComputeConfLossKernel<Dtype><<<CAFFE_GET_BLOCKS(num_threads),
    CAFFE_CUDA_NUM_THREADS>>>(num_threads, conf_gpu_data, num_preds_per_class,
        num_classes, loss_type, match_blob.gpu_data(), conf_loss_gpu_data);
  // Save the loss.
  all_conf_loss->clear();
  const Dtype* loss_data = conf_loss_blob.cpu_data();
  for (int i = 0; i < num; ++i) {
    vector<float> conf_loss(loss_data, loss_data + num_preds_per_class);
    all_conf_loss->push_back(conf_loss);
    loss_data += num_preds_per_class;
  }
}

// Explicit initialization.
template void ComputeConfLossGPU(const Blob<float>& conf_data, const int num,
      const int num_preds_per_class, const int num_classes,
      const int background_label_id, const ConfLossType loss_type,
      const vector<map<int, vector<int> > >& all_match_indices,
      const map<int, vector<NormalizedBBox> >& all_gt_bboxes,
      vector<vector<float> >* all_conf_loss);
template void ComputeConfLossGPU(const Blob<double>& conf_data, const int num,
      const int num_preds_per_class, const int num_classes,
      const int background_label_id, const ConfLossType loss_type,
      const vector<map<int, vector<int> > >& all_match_indices,
      const map<int, vector<NormalizedBBox> >& all_gt_bboxes,
      vector<vector<float> >* all_conf_loss);
	  
//********************************************************************	  
template <typename Dtype>
__global__ void ComputePboxConfLossKernel(const int nthreads,
    const Dtype* conf_data, const int num_preds_per_class,
    const int num_classes, const PConfLossType loss_type,
    const Dtype* match_data, Dtype* conf_loss_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int label = match_data[index];
    int num = index / num_preds_per_class;
    int p = index % num_preds_per_class;
    int start_idx = (num * num_preds_per_class + p) * num_classes;
    Dtype loss = 0;
    if (loss_type == MultiPBoxLossParameter_ConfLossType_SOFTMAX) {
      // Compute softmax probability.
      Dtype prob = conf_data[start_idx + label];
      loss = -log(Max(prob, Dtype(FLT_MIN)));
    } else if (loss_type == MultiPBoxLossParameter_ConfLossType_LOGISTIC) {
      int target = 0;
      for (int c = 0; c < num_classes; ++c) {
        if (c == label) {
          target = 1;
        } else {
          target = 0;
        }
        Dtype input = conf_data[start_idx + c];
        loss -= input * (target - (input >= 0)) -
          log(1 + exp(input - 2 * input * (input >= 0)));
      }
    }
    conf_loss_data[index] = loss;
  }
}


template <typename Dtype>
void ComputePboxConfLossGPU(const Blob<Dtype>& conf_blob, const int num,
      const int num_preds_per_class, const int num_classes,
      const int background_label_id, const PConfLossType loss_type,
      const vector<map<int, vector<int> > >& all_match_indices,
      const map<int, vector<NormalizedPBox> >& all_gt_pboxes,
      vector<vector<float> >* all_conf_loss) {
  CHECK_LT(background_label_id, num_classes);
  Blob<Dtype> match_blob(num, num_preds_per_class, 1, 1);
  Dtype* match_data = match_blob.mutable_cpu_data();
  for (int i = 0; i < num; ++i) {
    const map<int, vector<int> >& match_indices = all_match_indices[i];
    for (int p = 0; p < num_preds_per_class; ++p) {
      // Get the label index.
      int label = background_label_id;
      for (map<int, vector<int> >::const_iterator it =
           match_indices.begin(); it != match_indices.end(); ++it) {
        const vector<int>& match_index = it->second;
        CHECK_EQ(match_index.size(), num_preds_per_class);
        if (match_index[p] > -1) {
          CHECK(all_gt_pboxes.find(i) != all_gt_pboxes.end());
          const vector<NormalizedPBox>& gt_pboxes =
              all_gt_pboxes.find(i)->second;
          CHECK_LT(match_index[p], gt_pboxes.size());
          label = gt_pboxes[match_index[p]].label();
          CHECK_GE(label, 0);
          CHECK_NE(label, background_label_id);
          CHECK_LT(label, num_classes);
          // A prior can only be matched to one gt pbox.
          break;
        }
      }
      match_data[i * num_preds_per_class + p] = label;
    }
  }
  // Get probability data.
  const Dtype* conf_gpu_data = conf_blob.gpu_data();
  Blob<Dtype> prob_blob;
  prob_blob.ReshapeLike(conf_blob);
  if (loss_type == MultiPBoxLossParameter_ConfLossType_SOFTMAX) {
    Dtype* prob_gpu_data = prob_blob.mutable_gpu_data();
    SoftMaxGPU(conf_blob.gpu_data(), num * num_preds_per_class, num_classes, 1,
        prob_gpu_data);
    conf_gpu_data = prob_blob.gpu_data();
  }
  // Compute the loss.
  Blob<Dtype> conf_loss_blob(num, num_preds_per_class, 1, 1);
  Dtype* conf_loss_gpu_data = conf_loss_blob.mutable_gpu_data();
  const int num_threads = num * num_preds_per_class;
  // NOLINT_NEXT_LINE(whitespace/operators)
  ComputePboxConfLossKernel<Dtype><<<CAFFE_GET_BLOCKS(num_threads),
    CAFFE_CUDA_NUM_THREADS>>>(num_threads, conf_gpu_data, num_preds_per_class,
        num_classes, loss_type, match_blob.gpu_data(), conf_loss_gpu_data);
  // Save the loss.
  all_conf_loss->clear();
  const Dtype* loss_data = conf_loss_blob.cpu_data();
  for (int i = 0; i < num; ++i) {
    vector<float> conf_loss(loss_data, loss_data + num_preds_per_class);
    all_conf_loss->push_back(conf_loss);
    //LOG(WARNING) << "confloss " << conf_loss[1];
    loss_data += num_preds_per_class;
  }
}

// Explicit initialization.
template void ComputePboxConfLossGPU(const Blob<float>& conf_data, const int num,
      const int num_preds_per_class, const int num_classes,
      const int background_label_id, const PConfLossType loss_type,
      const vector<map<int, vector<int> > >& all_match_indices,
      const map<int, vector<NormalizedPBox> >& all_gt_pboxes,
      vector<vector<float> >* all_conf_loss);
template void ComputePboxConfLossGPU(const Blob<double>& conf_data, const int num,
      const int num_preds_per_class, const int num_classes,
      const int background_label_id, const PConfLossType loss_type,
      const vector<map<int, vector<int> > >& all_match_indices,
      const map<int, vector<NormalizedPBox> >& all_gt_pboxes,
      vector<vector<float> >* all_conf_loss);

}  // namespace caffe
